
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
/* Gives us high-resolution timers. */
#define _POSIX_C_SOURCE 199309L
#include <time.h>

/* OSX timer includes */
#ifdef __MACH__
  #include <mach/mach.h>
  #include <mach/mach_time.h>
#endif

#define N_IT 1
#define SUM_MAX 1e14
#define DTPB 512

//cpu function
void Allocate_Memory();
void Free_Memory();
void Send_To_Device();
void GPU_Compute();
void Send_To_Host();
void Save_Result();
void Prefix_sum(int *d_a, int *d_b);
void Bitonic_Sort(int *d_a, char flag);
void Find_Medroid();
void Label_Point();
char Judge();
static void print_time(double const seconds);
char Load_File(char *str);
void Calculate_2N();
void Initialize();



//cuda function (didn't fintout a way to split)

__global__ void Bitonic_Sort_Step(int *d_a, int j, int k, int N, char flag);
__global__ void Prefix_Up_Sweep(int *d_a, int N, int N_total);
__global__ void Prefix_Down_Sweep(int *d_a, int *d_b);
__global__ void Offest_Between_Block(int *d_b, int N);
__global__ void Add_Offset(int *d_a, int *d_b);
__global__ void Nearest_Medroid(double *x, int *label, int *clist, int *id, int *id_old, int *nlist, int Nij, int NC, int N, int N2, int Dim);
__global__ void Nlist_Initial(int *nlist, int *label, int N);
__global__ void Nlist_Reduction(int *nlist, int offset, int gap, int NC);
__global__ void Nlist_To_Prefix(int *nlist, int *prefix, int NC, int NC2, int BBPG, int BPG, int N);
__global__ void Distance_Between_Cluster(double *x, double *sum, int *label, int *prefix, int Dim, int Nij, int N, int N2);
__global__ void Min_Initial(double *min, double *sum, int *label, int *prefix, int *id, int NC, int N, int N2);
__global__ void Min_Reduction(double *min, int *id, int offset, int gap, int NC);
__global__ void Min_Further(double *min, int *id, int NC, int BBPG, int BPG);
__global__ void Update_ID(int *id, int*id_old, int NC, int *d_flag);
__device__ double Calculate_Distance(double *x, int i, int j, int Dim);


static inline double monotonic_seconds(){
#ifdef __MACH__
  /* OSX */
  static mach_timebase_info_data_t info;
  static double seconds_per_unit;
  if(seconds_per_unit == 0) {
    mach_timebase_info(&info);
    seconds_per_unit = (info.numer / info.denom) / 1e9;
  }
  return seconds_per_unit * mach_absolute_time();
#else
  /* Linux systems */
  struct timespec ts;
  clock_gettime(CLOCK_MONOTONIC, &ts);
  return ts.tv_sec + ts.tv_nsec * 1e-9;
#endif
}




int NC, NC2, TPB, BPG, BPG2, BPGC, BPGR, NP, NP2, Dim, Nij, Ncj;
int *h_label, *h_id, *h_id_old, *h_flag;
double *h_x;
int *d_label, *d_clist, *d_id, *d_id_old, *d_nlist, *d_prefix, *d_flag;
double *d_x, *d_min, *d_sum;

int main(int argc, char** argv){
	double st,et;
	if(argc<5) {
		printf("Not enough of arguemt\n");
		return 1;
	}
	if(argc>5) {
		printf("Too many arguements\n");
		return 2;
	}
	
	NC=atoi(argv[2]);
	TPB= atoi(argv[3]);
    BPG=atoi(argv[4]);
    
	if(Load_File(argv[1])){
		return 3;
	}

	st=monotonic_seconds();
	Initialize();
    Send_To_Device();
    GPU_Compute();
    Send_To_Host();
	et=monotonic_seconds();
	print_time(et-st);
	Save_Result();
	Free_Memory();
	return 0;
}



/**
* @brief Output the seconds elapsed while clustering.
*
* @param seconds Seconds spent on k-means clustering, excluding IO.
*/
static void print_time(double const seconds)
{
  printf("k-means clustering time: %0.04fs\n", seconds);
}

char Load_File(char *str){
	FILE *in;
	int i,j;
	in = fopen(str, "r");
	if(in == NULL){
		printf("can't find the file\n");
		return 1;
	}
	if(fscanf(in,"%d %d\n",&NP, &Dim )!=2){
		printf("can't get Number of nodes and relative dimension\n");
		return 2;
	}
    Calculate_2N();
	Allocate_Memory();
	for(i=0; i<NP; i++){
		for(j=0; j<Dim-1; j++){
			if(fscanf(in, "%f ", h_x+(j+Dim*i))<1){
				printf("can't get data\n");
				return 3;
			}
		}
		if(fscanf(in, "%f\n", h_x+((Dim-1)+Dim*i))<1){
			printf("can't get data\n");
			return 3;
		}
	}
		
	return 0;
}


void Calculate_2N(){
    
    Nij=(int)((NP+BPG*TPB-1)/(BPG*TPB));
    Ncj=(int)((NC+BPG*TPB-1)/(BPG*TPB));
    //fix for sorting, prefix sum and reduction
    NC2=0;
    NP2=1;
    while(NP2<NP){
        NP2<<=1;
    }
    while(NC2<NC){
        NC2+=DTPB;
    }
    BPG2=(int)((NP2+DTPB-1)/DTPB);
    BPGC=(int)(NC2/DTPB);
    BPGR=(int)((NP+DTPB-1)/DTPB);
    printf("NP=%d, NP2=%d, NC=%d, NC2=%d, Nij=%d, Ncj=%d, Dim=%d\n",NP, NP2, NC, NC2, Nij, Ncj,Dim);
    printf("TPB=%d, DTPB=%d, BPG=%d, BPG2=%d, BPGC=%d BPGR=%d\n",TPB, DTPB, BPG, BPG2, BPGC, BPGR);
}

void Initialize(){
    int i;
    for(i=0; i<NC; i++){
		h_id[i]=i;
		h_id_old[i]=i;
	}
}

void Save_Result(){
	FILE *out;
	int i,j;
	out = fopen("clusters.txt","w");
	for(i=0; i<NP; i++){
		fprintf(out, "%d\n",h_label[i]);
	}
	fclose(out);
	out = fopen("centroids.txt","w");
	for(i=0; i<NC; i++){
		for(j=0; j<Dim-1; j++){
			fprintf(out, "%lf ",h_x[j+h_id[i]*Dim]);
		}
		fprintf(out,"%lf\n",h_x[(Dim-1)+h_id[i]*Dim]);
	}
	fclose(out);
}


void Allocate_Memory(){   
    h_flag=(int*)malloc(BPG*sizeof(int));
    h_label=(int*)malloc(NP*sizeof(int));
    h_id=(int*)malloc(NC*sizeof(int));
    h_id_old=(int*)malloc(NC*sizeof(int));
    h_x=(double*)malloc(NP*Dim*sizeof(double));
    
     
    size_t size;

    size=NP*sizeof(int);
    hipMalloc((void**) &d_label, size);
    size=2*NP2*sizeof(int);
    hipMalloc((void**) &d_clist, size);
    //size=BPGR*NC*sizeof(int);
    size=NP*sizeof(int);
    hipMalloc((void**) &d_id, size);
    size=NC*BPG*sizeof(int);
    hipMalloc((void**) &d_id_old, size);
    size=BPGR*NC*sizeof(int);
    hipMalloc((void**) &d_nlist, size);
    size=(NC2+1)*sizeof(int);
    hipMalloc((void**) &d_prefix, size);
    size=BPG*sizeof(int);
    hipMalloc((void**) &d_flag, size);
    size=NP*Dim*sizeof(double);
    hipMalloc((void**) &d_x, size);
    size=NP*sizeof(double);
    hipMalloc((void**) &d_sum, size);
    size=BPGR*NC*sizeof(double);
    hipMalloc((void**) &d_min, size);
  
}
void Send_To_Device(){
    size_t size;
    size=NP*Dim*sizeof(double);
    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
    size=NC*sizeof(int);
    hipMemcpy(d_id, h_id, size, hipMemcpyHostToDevice);
    hipMemcpy(d_id_old, h_id_old, size, hipMemcpyHostToDevice);
}
void GPU_Compute(){
    char flag=1;
    int it=0;
    while((it<N_IT)&&flag){
        Label_Point();
        Find_Medroid();
    //    flag=Judge();
        it++;
    }
}

void Send_To_Host(){
    size_t size;

    size=NC*sizeof(int);
    hipMemcpy(h_id, d_id, size, hipMemcpyDeviceToHost);
    size=NP*sizeof(int);
    hipMemcpy(h_label, d_label, size, hipMemcpyDeviceToHost);

}
void Free_Memory(){
    free(h_label);
    free(h_id);
    free(h_id_old);
    free(h_x);

    hipFree(d_label);
    hipFree(d_id);
    hipFree(d_id_old);
    hipFree(d_nlist);
    hipFree(d_prefix);
    hipFree(d_flag);
    hipFree(d_x);
    hipFree(d_min);
    hipFree(d_sum);    

}

char Judge(){
    
    char flag=0;
    int i;
    size_t size=sizeof(int);
    //bind with reduction (TPB fix)
    Update_ID<<<BPGR,DTPB>>>(d_id, d_id_old, NC, d_flag);
    hipMemcpy(h_flag, d_flag, size, hipMemcpyDeviceToHost);
    for(i=0; i<BPG; i++){
        flag|=h_flag[i];
    }
    return flag;
}


void Prefix_sum(int *d_a, int *d_b){
    printf("TPB=%d, BPGC=%d, NC2=%d, N_total=%d\n",TPB, BPGC, NC2, NP);
    Prefix_Up_Sweep<<<BPGC,DTPB>>>(d_a, NC2, NP);
    Prefix_Down_Sweep<<<BPGC,DTPB>>>(d_a, d_b);
    if(BPG>1){
        Offest_Between_Block<<<1,1>>>(d_b, BPG);
        Add_Offset<<<BPGC, DTPB>>>(d_a, d_b);
    }
}

void Bitonic_Sort(int* d_a, char flag){
    int j,k;   
    for(k=2; k<=NP2; k<<=1){
        for(j=k>>1; j>0; j>>=1){
            Bitonic_Sort_Step<<<BPG2,DTPB>>>(d_a, j, k, NP2, flag);
        }
    }
}

void Label_Point(){
    int BBPG=(int)((BPGR+DTPB-1)/DTPB);
    int i, offset;

    Nearest_Medroid<<<BPG,TPB>>>(d_x, d_label, d_clist ,d_id, d_id_old, d_nlist, Nij, NC,NP, NP2,Dim);           
    //reduction for number of each cluster (fix TPB)
    Nlist_Initial<<<BPGR,DTPB>>>(d_nlist, d_label, NP);
    for(i=0; i<NC; i++){
        offset=i*BPGR;
        Nlist_Reduction<<<BBPG,DTPB>>>(d_nlist, offset, BPGR, NC);
    }
    //gather to prefix (fix TPB)
    Nlist_To_Prefix<<<BPGC,DTPB>>>(d_nlist, d_prefix, NC, NC2, BBPG, BPGR, NP);
    //prefix_sum (fix TPB)
    Prefix_sum(d_prefix, d_nlist);
    //sort (fix TPB)
    Bitonic_Sort(d_clist, 0);
}

void Find_Medroid(){
    int BBPG=(int)((BPGR+DTPB-1)/DTPB);
    int i, offset; 
    int size;
    double *h_temp;
    int *h_clist, *h_tempi;

    Distance_Between_Cluster<<<BPG, TPB>>>(d_x, d_sum, d_clist, d_prefix, Dim, Nij, NP, NP2);
    
    size=NP*sizeof(double);
    h_temp=(double*)malloc(size);
    size=2*NP2*sizeof(int);
    h_clist=(int*)malloc(size);
    h_tempi=(int*)malloc(size);

    //reduction for min (fix TPB)
    Min_Initial<<<BPGR,DTPB>>>(d_min, d_sum, d_clist, d_prefix, d_id, NC, NP, NP2);
    size=BPGR*NC*sizeof(int);
    hipMemcpy(h_tempi, d_id, size, hipMemcpyDeviceToHost);
    size=BPGR*NC*sizeof(double);
    hipMemcpy(h_temp, d_min, size, hipMemcpyDeviceToHost);
    printf("first\n");
    for(i=0; i<BPGR*NC; i++){
        printf("%d: %d %f\n",i, h_tempi[i], h_temp[i]);
        if(i%(BPGR)==BPGR-1){
            printf("=====================================\n");
        }
    }  


    for(i=0; i<NC; i++){
        offset=i*BPGR;
        Min_Reduction<<<BBPG,DTPB>>>(d_min, d_id, offset, BPGR, NC);
    }
    printf("\nsecond\n");
    size=BPGR*NC*sizeof(int);
    hipMemcpy(h_tempi, d_id, size, hipMemcpyDeviceToHost);
    size=BPGR*NC*sizeof(double);
    hipMemcpy(h_temp, d_min, size, hipMemcpyDeviceToHost);
    for(i=0; i<BPGR*NC; i++){
        printf("%d: %d %f\n",i, h_tempi[i], h_temp[i]);
        if(i%(BPGR)==BPGR-1){
            printf("=====================================\n");
        }
    }  


    Min_Further<<<BPGC,DTPB>>>(d_min, d_id, NC, BBPG, BPGR);

}




__global__ void Prefix_Up_Sweep(int *d_a, int N, int N_total){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
	int I = threadIdx.x;
    int l;

    __syncthreads();
    for(int stride=2; stride<=blockDim.x; stride<<=1){
        l=stride>>1;
        if((I&(stride-1))==0){
            d_a[i+stride-1]+=d_a[i+stride-1-l];
        }
        __syncthreads();
    }
    

}
__global__ void Prefix_Down_Sweep(int *d_a, int *d_b){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
	int I = threadIdx.x;
    int N2 = blockDim.x*(blockIdx.x+1);
    if(i<gridDim.x){
        d_b[i]=0;
    }
    __syncthreads();
    if(I==blockDim.x-1){
        d_b[blockIdx.x+1]+=d_a[i];
        d_a[i]=0;
    }
 
    __syncthreads();
    for(int stride=blockDim.x; stride>1; stride>>=1){
        int l=stride>>1;
        if((I&(stride-1))==0){
            int t=d_a[N2-1-I-l];
            d_a[N2-1-I-l]=d_a[N2-1-I];
            d_a[N2-1-I]+=t;
        }
        __syncthreads();
    }
}

__global__ void Offest_Between_Block(int *d_b, int N){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    //serial prefix scan
    if(i==0){
        d_b[0]=0;
        for(int i1=0; i1<N; i1++){
            d_b[i1+1]+=d_b[i1];
        }
    }
}

__global__ void Add_Offset(int *d_a, int *d_b){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    d_a[i]+=d_b[blockIdx.x];
}

__global__ void Bitonic_Sort_Step(int *d_a, int j, int k, int N, char flag){
    int i, ixj,m;
    i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i<N){
        m=i&k;
        if(flag){
            i+=N;
        }
        ixj=i^j;
        
        if((ixj>i)){
            if(m==0){
                if(d_a[i]>d_a[ixj]){
                    int temp=d_a[i];
                    d_a[i]=d_a[ixj];
                    d_a[ixj]=temp;
                    int i2=(i+N)&((N<<1)-1);
                    int ixj2=(ixj+N)&((N<<1)-1);
                    temp=d_a[i2];
                    d_a[i2]=d_a[ixj2];
                    d_a[ixj2]=temp;
                }
            }else{
                if(d_a[i]<d_a[ixj]){
                    int temp=d_a[i];
                    d_a[i]=d_a[ixj];
                    d_a[ixj]=temp;
                    int i2=(i+N)&((N<<1)-1);
                    int ixj2=(ixj+N)&((N<<1)-1);
                    temp=d_a[i2];
                    d_a[i2]=d_a[ixj2];
                    d_a[ixj2]=temp;
                }
            }
        }
    }
}

__global__ void Nearest_Medroid(double *x,int *label, int *clist,int *id, int *id_old, int *nlist, int Nij, int NC, int N, int N2, int Dim){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = Nij*i;
    double min2,temp;
    
    for(int j1=0; j1<Nij; j1++){
        if((j1+j)<NC*gridDim.x){
            nlist[j1+j]=0;
            id_old[j1+j]=id[(j1+j)%NC];
        }
    }
    __syncthreads();
    
    for(int j1=0; j1<Nij; j1++){
        if((j1+j)<N){
            min2=SUM_MAX;
            //initial label for array
            for(int i1=0; i1<NC; i1++){
                temp=Calculate_Distance(x,j+j1,id[i1],Dim);
                if(temp<min2){
                    label[j1+j]=i1;
                    min2=temp;
                }
            }
//            atomicAdd(&nlist[blockIdx.x+label[j1+j]*gridDim.x],1);
            clist[j1+j]=label[j1+j];
            clist[j1+j+N2]=j1+j;
        }
        if(j1+j<(N2-N)){
            //initial label for redundent
            clist[j1+j+N]=NC+2;
            clist[j1+j+N+N2]=N+1;
        }
    }
}

__global__ void Nlist_Initial(int *nlist, int *label, int N){
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if(i<N){
        atomicAdd(&nlist[blockIdx.x+label[i]*gridDim.x],1);
    }
}

__global__ void Nlist_Reduction(int *nlist, int offset, int gap, int NC){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int I = threadIdx.x;
    if(i<gap){
        for(int stride = blockDim.x>>1; stride>0; stride>>=1){
            if(I<stride){
                if(i+stride<gap){
                    nlist[i+offset]+=nlist[i+stride+offset];
                }
            }
            __syncthreads();
        }
    }
}

__global__ void Nlist_To_Prefix(int *nlist, int *prefix, int NC, int NC2, int BBPG,int BPG, int N){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i<NC2){
        prefix[i]=0;
    }
    if(i<NC){
        for(int i1=0; i1 < BBPG; i1++){
            prefix[i]+=nlist[i*BPG+i1*BBPG];
        }
    }
    if(i==0){
        prefix[NC2]=N;
    }
}

__global__ void Distance_Between_Cluster(double *x, double *sum, int *label, int *prefix, int Dim, int Nij, int N, int N2){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = i*Nij;
    for(int j1=0; j1<Nij; j1++){
        if((j1+j)<N){
            sum[j1+j]=0;
            for(int i1=prefix[label[j1+j]]; i1<prefix[label[j1+j]+1]; i1++ ){
                sum[j1+j]+=Calculate_Distance(x,label[j1+j+N2],label[i1+N2],Dim)/(prefix[label[j1+j]+1]-prefix[label[j1+j]]);
            }
        }
    }
}

__global__ void Min_Initial(double *min, double *sum, int *label, int *prefix, int *id, int NC, int N, int N2){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int I = threadIdx.x;
    int up,down;
    __shared__ double temp[1024];
    __shared__ int tag[1024];

    if(i<NC*gridDim.x){
        id[i]=N+2;
    }
    __syncthreads();
    up=0;
    down=-1;
    for(int i1=0; i1<NC; i1++){
        if((i-I)>=prefix[i1]){
            down++;
        }
        if((i-I+blockDim.x)>prefix[i1]){
            up++;
        }
    }

    //no atomic min function for double, therefore doing reduction inside a block for each cluster; 

    for(int i1=down; i1<up; i1++){
        if(i<N){
            if(label[i]==i1){
                tag[I]=label[i+N2];
                temp[I]=sum[i];
            }else{
                tag[I]=N+2;
                temp[I]=1e16;
            }         
            __syncthreads();
            for(int stride = blockDim.x>>1; stride>0; stride>>=1){
                if(I<stride){
                    if(i+stride<N){
                        if((tag[I]!=N+2)&&(tag[I+stride]!=N+2)){
                            if(temp[I+stride]<temp[I]){
                                temp[I]=temp[I+stride];
                                tag[I]=tag[I+stride];
                            }
                        }else if(tag[I+stride]!=N+2){
                            temp[I]=temp[I+stride];
                            tag[I]=tag[I+stride];
                        } 
                    }
                }
                __syncthreads();
            }

            if(I==0){
                min[blockIdx.x+i1*gridDim.x]=temp[0];
                id[blockIdx.x+i1*gridDim.x]=tag[0];
            }
          
        }
    }

}

__global__ void Min_Reduction(double *min, int *id, int offset, int gap, int NC){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int I = threadIdx.x;
    if(i<gap){
        for(int stride = blockDim.x>>1; stride>0; stride>>=1){
            if(I<stride){
                if(i+stride<gap){
                    if(min[i+offset+stride]<min[i+offset]){
                        min[i+offset]=min[i+offset+stride];
                        id[i+offset]=id[i+offset+stride];
                    }
                }
            }
            __syncthreads();
        }
    }
}

__global__ void Min_Further(double *min, int *id, int NC, int BBPG,int BPG){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i<NC){
        for(int i1=1; i1 < BBPG; i1++){
            if(min[i*BPG+i1*BBPG]<min[i]){
                min[i]=min[i*BPG+i1*BBPG];
                id[i]=id[i*BPG+i1*BBPG];
            }
        }
        __syncthreads();
        min[i]=min[i*BPG];
        id[i]=id[i*BPG];
    }
 }

 __global__ void Update_ID(int *id, int*id_old, int NC, int *d_flag){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i<gridDim.x){
        d_flag[i]=0;
    }
    __syncthreads();
    if(i<NC){
        if(id_old[i]!=id[i]){
            atomicAdd(&d_flag[blockIdx.x],1);
            id_old[i]=id[i];
        }
    }
 }

__device__ double Calculate_Distance(double *x, int i, int j, int Dim){
    double temp=0;
    int k=0;
    for(k=0; k<Dim; k++){
        temp+=(x[k+i*Dim]-x[k+j*Dim])*(x[k+i*Dim]-x[k+j*Dim]);
	}
	return sqrtf(temp);
}